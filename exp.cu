#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>

extern "C" {
    #include "sha.h"
}

void HexToBytes(const std::string& hex, unsigned char * &newsalt) {
    for (unsigned int i = 0; i < hex.length(); i += 2) {
      std::string byteString = hex.substr(i, 2);
      char byte = (char) strtol(byteString.c_str(), NULL, 16);
      newsalt[i/2] = byte;
    }
}

// Compile with: nvcc exp.cu sha384-512.c sha224-256.c sha1.c usha.c hmac.c -o build/exp

void pbkdf2(std::string password, std::string salt, uint8_t digest[USHAMaxHashSize]) {

    // Hashing function will be sha256. hlen will therefore be 32, same as keyLen.
    // Desired key length will be 32.
    // iterations will be 100000.
    int rounds = 100000;
    unsigned char * pw = (unsigned char *)password.c_str();
    int pwsize = password.size();

    uint8_t dk[USHAMaxHashSize];

    unsigned char * newsalt = (unsigned char *)malloc(16);
    HexToBytes(salt, newsalt);
    newsalt[16] = (1 >> 24) & 0xff;
    newsalt[17] = (1 >> 16) & 0xff;
    newsalt[18] = (1 >> 8) & 0xff;
    newsalt[19] = (1 >> 0) & 0xff;

    hmac(
        SHA256,
        newsalt,
        20,
        pw,
        pwsize,
        digest
    );

    for (int a = 0; a < 32; a++) {
        dk[a] = digest[a];
    }

    uint8_t newdigest[32];
    uint8_t runningkey[32];
    memcpy(runningkey, dk, 32);
    for (int i = 2; i <= rounds; i++) {
        hmac(
            SHA256,
            runningkey,
            32,
            pw,
            pwsize,
            newdigest
        );

        for (int j = 0; j < 32; j++) {
            dk[j] = dk[j] ^ newdigest[j];
            runningkey[j] = newdigest[j];
        }
    }

    for (int b = 0; b < 32; b++) {
        digest[b] = dk[b];
    }
}



int main(void)
{
    std::cout << "Running" << std::endl;

    std::string testpw = "glassy ubiquity absence";
    std::string testsalt = "2db485972861e63479528bf382d1bc04";
    std::string testhash = "3c453512d47b37352bf2c5c1408ea4d9f46c48878782843a685c0c7e54232ba0";

    unsigned char * newsalt = (unsigned char *)malloc(16);
    HexToBytes(testsalt, newsalt);

    uint8_t prk[USHAMaxHashSize];

    hmac(
        SHA256,
        newsalt,
        16,
        (unsigned char *)testpw.c_str(),
        testpw.size(),
        prk
    );

    //char * hex_str = "";
    //hex_str = itoa(*prk, hex_str, 16);
    //sprintf(hex_str.c_str(),"%x", *prk);
    
    for (int i = 0; i < SHA256HashSize; i++) {
        std::cout << std::setw(2) << std::setfill('0') << std::hex << static_cast<int>(prk[i]);
    }
    std::cout << std::endl;

    std::cout << "hmac Done" << std::endl;

    uint8_t pdprk[USHAMaxHashSize];

    pbkdf2(testpw, testsalt, pdprk);

    for (int i = 0; i < SHA256HashSize; i++) {
        //printf("%x", prk[i]);
        std::cout << std::setw(2) << std::setfill('0') << std::hex << static_cast<int>(pdprk[i]);
    }
    std::cout << std::endl;

    std::cout << "pbkdf2 Done" << std::endl;

    return 0;
}

