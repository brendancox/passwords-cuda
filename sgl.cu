#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <vector>
#include <fstream>
#include <chrono>

#include "sgl.h"


#define SHA_Ch(x,y,z)        (((x) & (y)) ^ ((~(x)) & (z)))
#define SHA_Maj(x,y,z)       (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))

/* Define the SHA shift, rotate left, and rotate right macros */
#define SHA256_SHR(bits,word)      ((word) >> (bits))
#define SHA256_ROTL(bits,word)                         \
  (((word) << (bits)) | ((word) >> (32-(bits))))
#define SHA256_ROTR(bits,word)                         \
  (((word) >> (bits)) | ((word) << (32-(bits))))

/* Define the SHA SIGMA and sigma macros */
#define SHA256_SIGMA0(word)   \
  (SHA256_ROTR( 2,word) ^ SHA256_ROTR(13,word) ^ SHA256_ROTR(22,word))
#define SHA256_SIGMA1(word)   \
  (SHA256_ROTR( 6,word) ^ SHA256_ROTR(11,word) ^ SHA256_ROTR(25,word))
#define SHA256_sigma0(word)   \
  (SHA256_ROTR( 7,word) ^ SHA256_ROTR(18,word) ^ SHA256_SHR( 3,word))
#define SHA256_sigma1(word)   \
  (SHA256_ROTR(17,word) ^ SHA256_ROTR(19,word) ^ SHA256_SHR(10,word))

/* Local Function Prototypes */
__host__ __device__ void SHA224_256ProcessMessageBlock(SHA256Context *context);
__host__ __device__  void SHA224_256Finalize(SHA256Context *context,
  uint8_t Pad_Byte);
__host__ __device__  void SHA224_256PadMessage(SHA256Context *context,
  uint8_t Pad_Byte);


// How many to run in parallel.
const int IN_PARALLEL = 256;

/*
 * SHA256Input
 *
 * Description:
 *   This function accepts an array of octets as the next portion
 *   of the message.
 *
 * Parameters:
 *   context: [in/out]
 *     The SHA context to update.
 *   message_array[ ]: [in]
 *     An array of octets representing the next portion of
 *     the message.
 *   length: [in]
 *     The length of the message in message_array.
 *
 * Returns:
 *   sha Error Code.
 */
 __host__ __device__ void SHA256Input(SHA256Context *context, const uint8_t *message_array,
    unsigned int length)
{
  while (length--) {
    context->Message_Block[context->Message_Block_Index++] =
            *message_array;

    uint32_t addTemp = context->Length_Low;
    if (((context->Length_Low += 8) < addTemp) && (++context->Length_High == 0)) {
      context->Corrupted = shaInputTooLong;
    }

    if ((context->Corrupted == shaSuccess) &&
      (context->Message_Block_Index == SHA256_Message_Block_Size))
      SHA224_256ProcessMessageBlock(context);

    message_array++;
  }
}


/*
 * SHA224_256Reset
 *
 * Description:
 *   This helper function will initialize the SHA256Context in
 *   preparation for computing a new SHA-224 or SHA-256 message digest.
 *
 * Parameters:
 *   context: [in/out]
 *     The context to reset.
 *   H0[ ]: [in]
 *     The initial hash value array to use.
 *
 * Returns:
 *   sha Error Code.
 */
 __host__ __device__ void SHA256Reset(SHA256Context *context)
{
  context->Length_High = context->Length_Low = 0;
  context->Message_Block_Index  = 0;

  context->Intermediate_Hash[0] = 0x6A09E667;
  context->Intermediate_Hash[1] = 0xBB67AE85;
  context->Intermediate_Hash[2] = 0x3C6EF372;
  context->Intermediate_Hash[3] = 0xA54FF53A;
  context->Intermediate_Hash[4] = 0x510E527F;
  context->Intermediate_Hash[5] = 0x9B05688C;
  context->Intermediate_Hash[6] = 0x1F83D9AB;
  context->Intermediate_Hash[7] = 0x5BE0CD19;

  context->Computed  = 0;
  context->Corrupted = shaSuccess;
}

/*
 * SHA224_256ProcessMessageBlock
 *
 * Description:
 *   This helper function will process the next 512 bits of the
 *   message stored in the Message_Block array.
 *
 * Parameters:
 *   context: [in/out]
 *     The SHA context to update.
 *
 * Returns:
 *   Nothing.
 *
 * Comments:
 *   Many of the variable names in this code, especially the
 *   single character names, were used because those were the
 *   names used in the Secure Hash Standard.
 */
 __host__ __device__ void SHA224_256ProcessMessageBlock(SHA256Context *context)
{
  /* Constants defined in FIPS 180-3, section 4.2.2 */
  static const uint32_t K[64] = {
      0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b,
      0x59f111f1, 0x923f82a4, 0xab1c5ed5, 0xd807aa98, 0x12835b01,
      0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7,
      0xc19bf174, 0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
      0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da, 0x983e5152,
      0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147,
      0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc,
      0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
      0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819,
      0xd6990624, 0xf40e3585, 0x106aa070, 0x19a4c116, 0x1e376c08,
      0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f,
      0x682e6ff3, 0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
      0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
  };
  int        t, t4;                   /* Loop counter */
  uint32_t   temp1, temp2;            /* Temporary word value */
  uint32_t   W[64];                   /* Word sequence */
  uint32_t   A, B, C, D, E, F, G, H;  /* Word buffers */

  /*
   * Initialize the first 16 words in the array W
   */
  for (t = t4 = 0; t < 16; t++, t4 += 4)
    W[t] = (((uint32_t)context->Message_Block[t4]) << 24) |
           (((uint32_t)context->Message_Block[t4 + 1]) << 16) |
           (((uint32_t)context->Message_Block[t4 + 2]) << 8) |
           (((uint32_t)context->Message_Block[t4 + 3]));

  for (t = 16; t < 64; t++)
    W[t] = SHA256_sigma1(W[t-2]) + W[t-7] +
        SHA256_sigma0(W[t-15]) + W[t-16];

  A = context->Intermediate_Hash[0];
  B = context->Intermediate_Hash[1];
  C = context->Intermediate_Hash[2];
  D = context->Intermediate_Hash[3];
  E = context->Intermediate_Hash[4];
  F = context->Intermediate_Hash[5];
  G = context->Intermediate_Hash[6];
  H = context->Intermediate_Hash[7];

  for (t = 0; t < 64; t++) {
    temp1 = H + SHA256_SIGMA1(E) + SHA_Ch(E,F,G) + K[t] + W[t];
    temp2 = SHA256_SIGMA0(A) + SHA_Maj(A,B,C);
    H = G;
    G = F;
    F = E;
    E = D + temp1;
    D = C;
    C = B;
    B = A;
    A = temp1 + temp2;
  }

  context->Intermediate_Hash[0] += A;
  context->Intermediate_Hash[1] += B;
  context->Intermediate_Hash[2] += C;
  context->Intermediate_Hash[3] += D;
  context->Intermediate_Hash[4] += E;
  context->Intermediate_Hash[5] += F;
  context->Intermediate_Hash[6] += G;
  context->Intermediate_Hash[7] += H;

  context->Message_Block_Index = 0;
}

/*
 * SHA224_256Finalize
 *
 * Description:
 *   This helper function finishes off the digest calculations.
 *
 * Parameters:
 *   context: [in/out]
 *     The SHA context to update.
 *   Pad_Byte: [in]
 *     The last byte to add to the message block before the 0-padding
 *     and length.  This will contain the last bits of the message
 *     followed by another single bit.  If the message was an
 *     exact multiple of 8-bits long, Pad_Byte will be 0x80.
 *
 * Returns:
 *   sha Error Code.
 */
 __host__ __device__ void SHA224_256Finalize(SHA256Context *context,
    uint8_t Pad_Byte)
{
  int i;
  SHA224_256PadMessage(context, Pad_Byte);
  /* message may be sensitive, so clear it out */
  for (i = 0; i < SHA256_Message_Block_Size; ++i)
    context->Message_Block[i] = 0;
  context->Length_High = 0;     /* and clear length */
  context->Length_Low = 0;
  context->Computed = 1;
}

/*
 * SHA224_256PadMessage
 *
 * Description:
 *   According to the standard, the message must be padded to the next
 *   even multiple of 512 bits.  The first padding bit must be a '1'.
 *   The last 64 bits represent the length of the original message.
 *   All bits in between should be 0.  This helper function will pad
 *   the message according to those rules by filling the
 *   Message_Block array accordingly.  When it returns, it can be
 *   assumed that the message digest has been computed.
 *
 * Parameters:
 *   context: [in/out]
 *     The context to pad.
 *   Pad_Byte: [in]
 *     The last byte to add to the message block before the 0-padding
 *     and length.  This will contain the last bits of the message
 *     followed by another single bit.  If the message was an
 *     exact multiple of 8-bits long, Pad_Byte will be 0x80.
 *
 * Returns:
 *   Nothing.
 */
 __host__ __device__ void SHA224_256PadMessage(SHA256Context *context,
    uint8_t Pad_Byte)
{

  /*
   * Check to see if the current message block is too small to hold
   * the initial padding bits and length.  If so, we will pad the
   * block, process it, and then continue padding into a second
   * block.
   */
  if (context->Message_Block_Index >= (SHA256_Message_Block_Size-8)) {
    context->Message_Block[context->Message_Block_Index++] = Pad_Byte;
    while (context->Message_Block_Index < SHA256_Message_Block_Size)
      context->Message_Block[context->Message_Block_Index++] = 0;
    SHA224_256ProcessMessageBlock(context);
  } else
    context->Message_Block[context->Message_Block_Index++] = Pad_Byte;

  while (context->Message_Block_Index < (SHA256_Message_Block_Size-8))
    context->Message_Block[context->Message_Block_Index++] = 0;

  /*
   * Store the message length as the last 8 octets
   */
  context->Message_Block[56] = (uint8_t)(context->Length_High >> 24);
  context->Message_Block[57] = (uint8_t)(context->Length_High >> 16);
  context->Message_Block[58] = (uint8_t)(context->Length_High >> 8);
  context->Message_Block[59] = (uint8_t)(context->Length_High);
  context->Message_Block[60] = (uint8_t)(context->Length_Low >> 24);
  context->Message_Block[61] = (uint8_t)(context->Length_Low >> 16);
  context->Message_Block[62] = (uint8_t)(context->Length_Low >> 8);
  context->Message_Block[63] = (uint8_t)(context->Length_Low);

  SHA224_256ProcessMessageBlock(context);
}

/*
 * SHA224_256ResultN
 *
 * Description:
 *   This helper function will return the 224-bit or 256-bit message
 *   digest into the Message_Digest array provided by the caller.
 *   NOTE:
 *    The first octet of hash is stored in the element with index 0,
 *    the last octet of hash in the element with index 27/31.
 *
 * Parameters:
 *   context: [in/out]
 *     The context to use to calculate the SHA hash.
 *   Message_Digest[ ]: [out]
 *     Where the digest is returned.
 *   HashSize: [in]
 *     The size of the hash, either 28 or 32.
 *
 * Returns:
 *   sha Error Code.
 */
 __host__ __device__ void SHA256Result(SHA256Context *context,
    uint8_t Message_Digest[SHA256HashSize])
{
  int i;
  SHA224_256Finalize(context, 0x80);

  for (i = 0; i < SHA256HashSize; ++i)
    Message_Digest[i] = (uint8_t)
      (context->Intermediate_Hash[i>>2] >> 8 * ( 3 - ( i & 0x03 ) ));
}


__host__ __device__ void hmac_combined(
  const unsigned char *message_array, int length,
  const unsigned char *key, int key_len,
  uint8_t digest[SHA256HashSize])
{
  int i;

  unsigned char k_ipad[SHA256_Message_Block_Size];
  unsigned char k_opad[SHA256_Message_Block_Size];

  for (i = 0; i < key_len; i++) {
    k_ipad[i] = key[i] ^ 0x36;
    k_opad[i] = key[i] ^ 0x5c;
  }
  for ( ; i < SHA256_Message_Block_Size; i++) {
    k_ipad[i] = 0x36;
    k_opad[i] = 0x5c;
  }

  SHA256Context shaContext;
  SHA256Context * context = &shaContext; 

  // Reset
  context->Length_High = context->Length_Low = 0;
  context->Message_Block_Index  = 0;
  context->Intermediate_Hash[0] = 0x6A09E667;
  context->Intermediate_Hash[1] = 0xBB67AE85;
  context->Intermediate_Hash[2] = 0x3C6EF372;
  context->Intermediate_Hash[3] = 0xA54FF53A;
  context->Intermediate_Hash[4] = 0x510E527F;
  context->Intermediate_Hash[5] = 0x9B05688C;
  context->Intermediate_Hash[6] = 0x1F83D9AB;
  context->Intermediate_Hash[7] = 0x5BE0CD19;
  context->Computed  = 0;
  context->Corrupted = shaSuccess;

  //SHA256Input(&shaContext, k_ipad, SHA256_Message_Block_Size);
  for (i = 0; i < SHA256_Message_Block_Size; i++) {
    context->Message_Block[context->Message_Block_Index++] = k_ipad[i];

    uint32_t addTemp = context->Length_Low;
    if (((context->Length_Low += 8) < addTemp) && (++context->Length_High == 0)) {
      context->Corrupted = shaInputTooLong;
    }

    if ((context->Corrupted == shaSuccess) &&
      (context->Message_Block_Index == SHA256_Message_Block_Size))
      SHA224_256ProcessMessageBlock(context);
  }

  //SHA256Input(&shaContext, message_array, length);
  for (i = 0; i < length; i++) {
    context->Message_Block[context->Message_Block_Index++] = message_array[i];

    uint32_t addTemp = context->Length_Low;
    if (((context->Length_Low += 8) < addTemp) && (++context->Length_High == 0)) {
      context->Corrupted = shaInputTooLong;
    }

    if ((context->Corrupted == shaSuccess) &&
      (context->Message_Block_Index == SHA256_Message_Block_Size))
      SHA224_256ProcessMessageBlock(context);
  }

  // Result
  SHA224_256Finalize(context, 0x80);
  for (i = 0; i < SHA256HashSize; ++i) {
    digest[i] = (uint8_t)(context->Intermediate_Hash[i>>2] >> 8 * ( 3 - ( i & 0x03 ) ));
  }

  // Reset
  context->Length_High = context->Length_Low = 0;
  context->Message_Block_Index  = 0;
  context->Intermediate_Hash[0] = 0x6A09E667;
  context->Intermediate_Hash[1] = 0xBB67AE85;
  context->Intermediate_Hash[2] = 0x3C6EF372;
  context->Intermediate_Hash[3] = 0xA54FF53A;
  context->Intermediate_Hash[4] = 0x510E527F;
  context->Intermediate_Hash[5] = 0x9B05688C;
  context->Intermediate_Hash[6] = 0x1F83D9AB;
  context->Intermediate_Hash[7] = 0x5BE0CD19;
  context->Computed  = 0;
  context->Corrupted = shaSuccess;

  //SHA256Input(&shaContext, k_opad, SHA256_Message_Block_Size);
  for (i = 0; i < SHA256_Message_Block_Size; i++) {
    context->Message_Block[context->Message_Block_Index++] = k_opad[i];

    uint32_t addTemp = context->Length_Low;
    if (((context->Length_Low += 8) < addTemp) && (++context->Length_High == 0)) {
      context->Corrupted = shaInputTooLong;
    }

    if ((context->Corrupted == shaSuccess) &&
      (context->Message_Block_Index == SHA256_Message_Block_Size))
      SHA224_256ProcessMessageBlock(context);
  }

  //SHA256Input(&shaContext, digest, SHA256HashSize);
  for (i = 0; i < SHA256HashSize; i++) {
    context->Message_Block[context->Message_Block_Index++] = digest[i];

    uint32_t addTemp = context->Length_Low;
    if (((context->Length_Low += 8) < addTemp) && (++context->Length_High == 0)) {
      context->Corrupted = shaInputTooLong;
    }

    if ((context->Corrupted == shaSuccess) &&
      (context->Message_Block_Index == SHA256_Message_Block_Size))
      SHA224_256ProcessMessageBlock(context);
  }

  // Result
  SHA224_256Finalize(context, 0x80);
  for (i = 0; i < SHA256HashSize; ++i) {
    digest[i] = (uint8_t)(context->Intermediate_Hash[i>>2] >> 8 * ( 3 - ( i & 0x03 ) ));
  }
}


__host__ __device__ void hmac(
    const unsigned char *message_array, int length,
    const unsigned char *key, int key_len,
    uint8_t digest[SHA256HashSize])
{
  int i;

  /* inner padding - key XORd with ipad */
  unsigned char k_ipad[SHA256_Message_Block_Size];

  /*
   * The HMAC transform looks like:
   *
   * SHA(K XOR opad, SHA(K XOR ipad, text))
   *
   * where K is an n byte key, 0-padded to a total of blocksize bytes,
   * ipad is the byte 0x36 repeated blocksize times,
   * opad is the byte 0x5c repeated blocksize times,
   * and text is the data being protected.
   */

   unsigned char k_opad[SHA256_Message_Block_Size];

  /* store key into the pads, XOR'd with ipad and opad values */
  for (i = 0; i < key_len; i++) {
    k_ipad[i] = key[i] ^ 0x36;
    k_opad[i] = key[i] ^ 0x5c;
  }
  /* remaining pad bytes are '\0' XOR'd with ipad and opad values */
  for ( ; i < SHA256_Message_Block_Size; i++) {
    k_ipad[i] = 0x36;
    k_opad[i] = 0x5c;
  }

  SHA256Context shaContext; 

  /* perform inner hash */
  /* init context for 1st pass */
  SHA256Reset(&shaContext);
  /* and start with inner pad */
  SHA256Input(&shaContext, k_ipad, SHA256_Message_Block_Size);

  // Run on the message array.
  SHA256Input(&shaContext, message_array, length);
  
  SHA256Result(&shaContext, digest);
  /* perform outer SHA */
  /* init context for 2nd pass */
  SHA256Reset(&shaContext);
  /* start with outer pad */
  SHA256Input(&shaContext, k_opad, SHA256_Message_Block_Size);
  /* then results of 1st hash */
  SHA256Input(&shaContext, digest, SHA256HashSize);
  /* finish up 2nd pass */
  SHA256Result(&shaContext, digest);
}

void HexToBytes(const std::string& hex, unsigned char * &newsalt) {
    for (unsigned int i = 0; i < hex.length(); i += 2) {
      std::string byteString = hex.substr(i, 2);
      char byte = (char) strtol(byteString.c_str(), NULL, 16);
      newsalt[i/2] = byte;
    }
}

// Compile with: nvcc sgl.cu -o build/sgl

__host__ __device__ void pbkdf2(unsigned char * password, int pwsize, unsigned char * salt, uint8_t digest[SHA256HashSize]) {

    // Hashing function will be sha256. hlen will therefore be 32, same as keyLen.
    // Desired key length will be 32.
    // iterations will be 100000.
    int rounds = 100000;

    hmac(
        salt,
        20,
        password,
        pwsize,
        digest
    );

    uint8_t newdigest[32];
    uint8_t runningkey[32];

    memcpy(runningkey, digest, 32);
    /*for (int i = 0; i < 32; i++) {
      runningkey[i] = newdigest[i];
    }*/

    for (int i = 2; i <= rounds; i++) {
        //hmac(runningkey, 32, password, pwsize, newdigest);
        hmac_combined(runningkey, 32, password, pwsize, newdigest);

        for (int j = 0; j < 32; j++) {
            digest[j] = digest[j] ^ newdigest[j];
            runningkey[j] = newdigest[j];
        }
    }
}

void createPbkdfSalt(unsigned char* newsalt, std::string salt) {
  HexToBytes(salt, newsalt);
  newsalt[16] = (1 >> 24) & 0xff;
  newsalt[17] = (1 >> 16) & 0xff;
  newsalt[18] = (1 >> 8) & 0xff;
  newsalt[19] = (1 >> 0) & 0xff;
}

void runIteration(std::string words[18328], unsigned char * salt, unsigned char * expected) {
  int rand1 = rand() % 18327;
  int rand2 = rand() % 18327;
  int rand3 = rand() % 18327;
  std::string password = words[rand1] + " " + words[rand2] + " " + words[rand3];

  uint8_t result[SHA256HashSize];

  pbkdf2((unsigned char *)password.c_str(), password.size(), salt, result);
  
  bool match = true;
  for (int j = 0; j < SHA256HashSize; j++) {
    if (result[j] != expected[j]) {
      match = false;
      break;
    }
  }

  if (match) {
    std::cout << "MATCH!!!: " << password << std::endl;
  }
}


__global__
void runIterationKernel(unsigned char* passwords, int * pwsizes, unsigned char * salt, unsigned char * expected, bool matches[IN_PARALLEL]) {

  uint8_t result[SHA256HashSize];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < IN_PARALLEL; i += stride) {
    unsigned char * password;
    password = passwords + i * 40;
    pbkdf2(password, pwsizes[i], salt, result);
    
    bool match = true;
    for (int j = 0; j < SHA256HashSize; j++) {
      if (result[j] != expected[j]) {
        match = false;
        break;
      }
    }

    if (match) {
      matches[i] = true;
    }
  }
}

void runInParallel() {

  std::cout << "Setting up parallel run" << std::endl;

  std::string words[18328];

  std::string line;
  std::ifstream myfile;
  myfile.open ("AgileWords.txt");
  if (myfile.is_open())
  {
      int i = 0;
    while ( getline (myfile,line) )
    {
      words[i] = line;
      i++;
    }
    myfile.close();
  }

  // ID: DOHB6DC7 -- overwritten for testing !!!!!
  //std::string saltstring = "9dc661ec09c948dd16710439d157cef2";
  std::string saltstring = "2db485972861e63479528bf382d1bc04";
  std::string expected = "3c453512d47b37352bf2c5c1408ea4d9f46c48878782843a685c0c7e54232ba0";
  //std::string expected = "4073c5e1cbd7790347b26e0447795220cd933689219b3446da294f509a583d48";



  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);
  std::cout << properties.name << std::endl;
  std::cout << "Threads per block: " << properties.maxThreadsPerBlock << std::endl;

  auto started = std::chrono::high_resolution_clock::now();

  std::string originals[IN_PARALLEL];
  unsigned char * passwords;
  int *pwsizes;
  bool *matches;
  unsigned char * salt;
  unsigned char * expectedBytes;
  hipMallocManaged(&passwords, IN_PARALLEL * 40 * IN_PARALLEL * sizeof(char));
  hipMallocManaged(&pwsizes, IN_PARALLEL * sizeof(int));
  hipMallocManaged(&matches, IN_PARALLEL * sizeof(bool));
  hipMallocManaged(&salt, 20 * sizeof(char));
  hipMallocManaged(&expectedBytes, 32 * sizeof(char));

  createPbkdfSalt(salt, saltstring);
  HexToBytes(expected, expectedBytes);

  for (int i = 0; i < IN_PARALLEL; i++) {

    if (i == 16) {
      originals[i] = "glassy ubiquity absence";

    } else {
      int rand1 = rand() % 18327;
      int rand2 = rand() % 18327;
      int rand3 = rand() % 18327;
      originals[i] = words[rand1] + " " + words[rand2] + " " + words[rand3];
      
    }

    pwsizes[i] = originals[i].size();
    unsigned char * password = (unsigned char *)originals[i].c_str();
    for (int j = 0; j < 40; j++) {
      if (j < pwsizes[i]) {
        passwords[i*40 + j] = password[j];
      } else {
        passwords[i*40 + j] = 0x00000000;
      }
      
    }
    matches[i] = false;
  }

  unsigned char * password = passwords + 16 * 40;

  for (int i = 0; i < pwsizes[16]; i++) {
    std::cout << password[i];
  }
  std::cout << std::endl;

  hipError_t error;

  int numblocks = 4;
  int blocksize = IN_PARALLEL / numblocks;

  runIterationKernel<<<numblocks, blocksize>>>(passwords, pwsizes, salt, expectedBytes, matches);
  std::cout << "Running parallel" << std::endl;
  hipDeviceSynchronize();
  error = hipGetLastError();
  std::cout << hipGetErrorName(error) << ": " << hipGetErrorString(error) << std::endl;
  std::cout << "Synchronized" << std::endl;

  for (int k = 0; k < IN_PARALLEL; k++) {
    if (matches[k]) {
      std::cout << "MATCH!!!: " << originals[k] << std::endl;
    } else {
      //std::cout << "Did not match: " << originals[k] << std::endl;
    }
  }

  hipFree(passwords);
  hipFree(pwsizes);
  hipFree(matches);
  hipFree(salt);
  hipFree(expectedBytes);

  auto done = std::chrono::high_resolution_clock::now();
  double totalTime = std::chrono::duration_cast<std::chrono::milliseconds>(done-started).count();
  totalTime = totalTime / 1000;
  std::cout << "Total time taken: " << std::fixed << totalTime << "s" << std::endl;

	// Number of combinations = 6,156,660,800,000
	// if 5/sec is 39,000 years
	// if 1000 takes 1min30sec then 17570
	// if 10000 takes 5min then 5856 years
  // And now with parallel cuda
  // about 1500/sec? = 130 years...
}

void loadWords() {
    std::string words[18328];

    std::string line;
    std::ifstream myfile;
    myfile.open ("AgileWords.txt");
    if (myfile.is_open())
    {
        int i = 0;
      while ( getline (myfile,line) )
      {
        words[i] = line;
        i++;
      }
      myfile.close();
    }

    std::cout << "Words loaded" <<std::endl;

    // ID: DOHB6DC7
    std::string saltstring = "9dc661ec09c948dd16710439d157cef2";
    unsigned char * salt = (unsigned char *)malloc(20);
    createPbkdfSalt(salt, saltstring);
    std::string expected = "4073c5e1cbd7790347b26e0447795220cd933689219b3446da294f509a583d48";
    unsigned char * expectedBytes = (unsigned char *)malloc(32);
    HexToBytes(expected, expectedBytes);

    int attempts = 10;

    std::cout << "About to start loop" <<std::endl;

    auto started = std::chrono::high_resolution_clock::now();

    for (int i = 0; i < attempts; i++) {
      runIteration(words, salt, expectedBytes);
    }

    auto done = std::chrono::high_resolution_clock::now();

    std::cout << "Loop done" <<std::endl;

    double totalTime = std::chrono::duration_cast<std::chrono::milliseconds>(done-started).count();
    totalTime = totalTime / 1000;

    std::cout << "Total time taken: " << std::fixed << totalTime << "s" << std::endl;
}



__global__
void increase(int n, int *x, bool *b)
{
  for (int i = 0; i < n; i++) {
    if (b[i]) {
      x[i] = x[i] + 20;
    }
  }
}

void testCuda() {
  
  int N = 5;
  int *x;
  bool *b;

  hipMallocManaged(&x, N*sizeof(int));
  hipMallocManaged(&b, N*sizeof(bool));

  for (int i = 0; i < N; i++) {
    x[i] = i;
    if (i % 3 == 0) {
      b[i] = false;
    } else {
      b[i] = true;
    }
  }

  increase<<<1,1>>>(N, x, b);

  hipDeviceSynchronize();

  for (int i = 0; i < N; i++) {
    std::cout << std::dec << x[i] << std::endl;
  }

  hipFree(x);
  hipFree(b);
}


int main(void)
{
    std::cout << "Running" << std::endl;

    std::string testpw = "glassy ubiquity absence";
    std::string testsalt = "2db485972861e63479528bf382d1bc04";
    std::string testhash = "3c453512d47b37352bf2c5c1408ea4d9f46c48878782843a685c0c7e54232ba0";

    unsigned char * newsalt = (unsigned char *)malloc(20);
    createPbkdfSalt(newsalt, testsalt);

    uint8_t prk[SHA256HashSize];

    hmac(
        newsalt,
        16,
        (unsigned char *)testpw.c_str(),
        testpw.size(),
        prk
    );
    
    for (int i = 0; i < SHA256HashSize; i++) {
        std::cout << std::setw(2) << std::setfill('0') << std::hex << static_cast<int>(prk[i]);
    }
    std::cout << std::endl;

    std::cout << "hmac Done" << std::endl;

    uint8_t pdprk[SHA256HashSize];

    pbkdf2((unsigned char *)testpw.c_str(), testpw.size(), newsalt, pdprk);

    for (int i = 0; i < SHA256HashSize; i++) {
        //printf("%x", prk[i]);
        std::cout << std::setw(2) << std::setfill('0') << std::hex << static_cast<int>(pdprk[i]);
    }
    std::cout << std::endl;

    unsigned char * expectedBytes = (unsigned char *)malloc(32);
    HexToBytes(testhash, expectedBytes);
    bool match = true;
    for (int j = 0; j < SHA256HashSize; j++) {
      if (pdprk[j] != expectedBytes[j]) {
        match = false;
        break;
      }
    }

    if (match) {
      std::cout << "pbkdf2 Test hash matched" << std::endl;
    }

    std::cout << "pbkdf2 Done" << std::endl;


    // The cracking..
    //loadWords();
    testCuda();

    runInParallel();

    return 0;
}

